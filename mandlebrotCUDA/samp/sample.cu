#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

#define BLOCK_SIZE 16

__global__ void printGPU(const unsigned char *text)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y == 0 && x == 0)
    {
        printf("%s", text);
    }
}

int main(void)
{
    char h_text[] = "Hello from GPU!\n";

    unsigned char *d_text;
    checkCudaErrors(hipMalloc(&d_text, sizeof(h_text)));

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(ceil(1.0 / blockSize.x), ceil(1.0 / blockSize.y));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    checkCudaErrors(hipMemcpy(d_text, h_text, sizeof(h_text), hipMemcpyHostToDevice));

    printGPU<<<gridSize, blockSize>>>(d_text);
    getLastCudaError("printGPU() execution failed\n");

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel & Memcpy Execution time is: %0.3f milliseconds \n", milliseconds);

    checkCudaErrors(hipFree(d_text));
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
